#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>

// ----------------------------------------------------------------------------
// kernel helper
// ----------------------------------------------------------------------------
template <typename T>
__global__ void k_set(T* ptr, size_t N, T value) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) {
    ptr[i] = value;
  }
}

template <typename T>
__global__ void k_single_set(T* ptr, int i, T value) {
  ptr[i] = value;
}

template <typename T>
__global__ void k_add(T* ptr, size_t N, T value) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) {
    ptr[i] += value;
  }
}

template <typename T>
__global__ void k_single_add(T* ptr, int i, T value) {
  ptr[i] += value;
}

// --------------------------------------------------------
// Testcase: Builder
// --------------------------------------------------------
TEST_CASE("Builder" * doctest::timeout(300)) {

  tf::cudaGraph G;
  tf::cudaFlow cf(G, [](){ return true; });

  int source = 1;
  int target = 1;

  auto copy1 = cf.copy(&target, &source, 1).name("copy1");
  auto copy2 = cf.copy(&target, &source, 1).name("copy2");
  auto copy3 = cf.copy(&target, &source, 1).name("copy3");

  REQUIRE(copy1.name() == "copy1");
  REQUIRE(copy2.name() == "copy2");
  REQUIRE(copy3.name() == "copy3");

  REQUIRE(!copy1.empty());
  REQUIRE(!copy2.empty());
  REQUIRE(!copy3.empty());
  
  copy1.precede(copy2);
  copy2.succeed(copy3);

  REQUIRE(copy1.num_successors() == 1);
  REQUIRE(copy2.num_successors() == 0);
  REQUIRE(copy3.num_successors() == 1);
}

// --------------------------------------------------------
// Testcase: Empty
// --------------------------------------------------------

TEST_CASE("Empty" * doctest::timeout(300)) {

  std::atomic<int> counter{0};
  
  tf::Taskflow taskflow;
  tf::Executor executor;

  taskflow.emplace([&](tf::cudaFlow&){ 
    ++counter; 
  });
  
  taskflow.emplace([&](tf::cudaFlow&){ 
    ++counter; 
  });
  
  taskflow.emplace([&](tf::cudaFlow&){ 
    ++counter; 
  });

  executor.run_n(taskflow, 100).wait();

  REQUIRE(counter == 300);
}

// --------------------------------------------------------
// Testcase: Set
// --------------------------------------------------------
template <typename T>
void set() {

  for(unsigned n=1; n<=123456; n = n*2 + 1) {

    tf::Taskflow taskflow;
    tf::Executor executor;
    
    T* cpu = nullptr;
    T* gpu = nullptr;

    auto cputask = taskflow.emplace([&](){
      cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
    });

    auto gputask = taskflow.emplace([&](tf::cudaFlow& cf) {
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel(g, b, 0, k_set<T>, gpu, n, (T)17);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == (T)17);
    }

    std::free(cpu);
    REQUIRE(hipFree(gpu) == hipSuccess);
  }
}

TEST_CASE("Set.i8" * doctest::timeout(300)) {
  set<int8_t>();
}

TEST_CASE("Set.i16" * doctest::timeout(300)) {
  set<int16_t>();
}

TEST_CASE("Set.i32" * doctest::timeout(300)) {
  set<int32_t>();
}

// --------------------------------------------------------
// Testcase: Add
// --------------------------------------------------------
template <typename T>
void add() {

  for(unsigned n=1; n<=123456; n = n*2 + 1) {
   
    tf::Taskflow taskflow;
    tf::Executor executor;
    
    T* cpu = nullptr;
    T* gpu = nullptr;
    
    auto cputask = taskflow.emplace([&](){
      cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
    });
    
    auto gputask = taskflow.emplace([&](tf::cudaFlow& cf){
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto ad1 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 1);
      auto ad2 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 2);
      auto ad3 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 3);
      auto ad4 = cf.kernel(g, b, 0, k_add<T>, gpu, n, 4);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(ad1);
      ad1.precede(ad2);
      ad2.precede(ad3);
      ad3.precede(ad4);
      ad4.precede(d2h);
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == 10);
    }

    std::free(cpu);
    REQUIRE(hipFree(gpu) == hipSuccess);
  }
}

TEST_CASE("Add.i8" * doctest::timeout(300)) {
  add<int8_t>();
}

TEST_CASE("Add.i16" * doctest::timeout(300)) {
  add<int16_t>();
}

TEST_CASE("Add.i32" * doctest::timeout(300)) {
  add<int32_t>();
}

// TODO: 64-bit fail?
//TEST_CASE("Add.i64" * doctest::timeout(300)) {
//  add<int64_t>();
//}


// --------------------------------------------------------
// Testcase: Binary Set
// --------------------------------------------------------
template <typename T>
void bset() {

  const unsigned n = 10000;

  tf::Taskflow taskflow;
  tf::Executor executor;

  T* cpu = nullptr;
  T* gpu = nullptr;
  
  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&](tf::cudaFlow& cf) {
    dim3 g = {1, 1, 1};
    dim3 b = {1, 1, 1};
    auto h2d = cf.copy(gpu, cpu, n);
    auto d2h = cf.copy(cpu, gpu, n);

    std::vector<tf::cudaTask> tasks(n+1);

    for(unsigned i=1; i<=n; ++i) {
      tasks[i] = cf.kernel(g, b, 0, k_single_set<T>, gpu, i-1, (T)17);

      auto p = i/2;
      if(p != 0) {
        tasks[p].precede(tasks[i]);
      }

      tasks[i].precede(d2h);
      h2d.precede(tasks[i]);
    }
  });

  cputask.precede(gputask);
  
  executor.run(taskflow).wait();

  for(unsigned i=0; i<n; ++i) {
    REQUIRE(cpu[i] == (T)17);
  }

  std::free(cpu);
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("BSet.i8" * doctest::timeout(300)) {
  bset<int8_t>();
}

TEST_CASE("BSet.i16" * doctest::timeout(300)) {
  bset<int16_t>();
}

TEST_CASE("BSet.i32" * doctest::timeout(300)) {
  bset<int32_t>();
}

// --------------------------------------------------------
// Testcase: Memset
// --------------------------------------------------------
TEST_CASE("Memset") {
  
  tf::Taskflow taskflow;
  tf::Executor executor;
  
  const int N = 100;

  int* cpu = new int [N];
  int* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(int)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = 999;
    }
    
    taskflow.emplace([&](tf::cudaFlow& cf){
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<int>, gpu, N, 123);
      auto zero = cf.memset(gpu+start, 0x3f, (N-start)*sizeof(int));
      auto copy = cf.copy(cpu, gpu, N);
      kset.precede(zero);
      zero.precede(copy);
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(cpu[i] == 123);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(cpu[i] == 0x3f3f3f3f);
    }
  }
  
  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

// --------------------------------------------------------
// Testcase: Memset0
// --------------------------------------------------------
template <typename T>
void memset0() {
  
  tf::Taskflow taskflow;
  tf::Executor executor;
  
  const int N = 97;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](tf::cudaFlow& cf){
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto zero = cf.memset(gpu+start, (T)0, (N-start)*sizeof(T));
      auto copy = cf.copy(cpu, gpu, N);
      kset.precede(zero);
      zero.precede(copy);
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)0) < 1e-4);
    }
  }
  
  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Memset0.i8") {
  memset0<int8_t>();
}

TEST_CASE("Memset0.i16") {
  memset0<int16_t>();
}

TEST_CASE("Memset0.i32") {
  memset0<int32_t>();
}

TEST_CASE("Memset0.f32") {
  memset0<float>();
}

TEST_CASE("Memset0.f64") {
  memset0<double>();
}

// --------------------------------------------------------
// Testcase: Memcpy
// --------------------------------------------------------
template <typename T>
void memcpy() {
  
  tf::Taskflow taskflow;
  tf::Executor executor;
  
  const int N = 97;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](tf::cudaFlow& cf){
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto zero = cf.memset(gpu+start, (T)0, (N-start)*sizeof(T));
      auto copy = cf.memcpy(cpu, gpu, N*sizeof(T));
      kset.precede(zero);
      zero.precede(copy);
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)0) < 1e-4);
    }
  }
  
  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Memcpy.i8") {
  memcpy<int8_t>();
}

TEST_CASE("Memcpy.i16") {
  memcpy<int16_t>();
}

TEST_CASE("Memcpy.i32") {
  memcpy<int32_t>();
}

TEST_CASE("Memcpy.f32") {
  memcpy<float>();
}

TEST_CASE("Memcpy.f64") {
  memcpy<double>();
}

// --------------------------------------------------------
// Testcase: fill
// --------------------------------------------------------
template <typename T>
void fill(T value) {
  
  tf::Taskflow taskflow;
  tf::Executor executor;
  
  const int N = 107;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](tf::cudaFlow& cf){
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto fill = cf.fill(gpu+start, value, (N-start));
      auto copy = cf.copy(cpu, gpu, N);
      kset.precede(fill);
      fill.precede(copy);
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - value) < 1e-4);
    }
  }

  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Fill.i8") {
  fill<int8_t>(+123);
  fill<int8_t>(-123);
}

TEST_CASE("Fill.i16") {
  fill<int16_t>(+12345);
  fill<int16_t>(-12345);
}

TEST_CASE("Fill.i32") {
  fill<int32_t>(+123456789);
  fill<int32_t>(-123456789);
}

TEST_CASE("Fill.f32") {
  fill<float>(+123456789.0f);
  fill<float>(-123456789.0f);
}

// --------------------------------------------------------
// Testcase: Zero
// --------------------------------------------------------
template <typename T>
void zero() {
  
  tf::Taskflow taskflow;
  tf::Executor executor;
  
  const int N = 100;

  T* cpu = new T [N];
  T* gpu = nullptr;
    
  REQUIRE(hipMalloc(&gpu, N*sizeof(T)) == hipSuccess);

  for(int r=1; r<=100; ++r) {

    int start = ::rand() % N;

    for(int i=0; i<N; ++i) {
      cpu[i] = (T)999;
    }
    
    taskflow.emplace([&](tf::cudaFlow& cf){
      dim3 g = {(unsigned)(N+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto kset = cf.kernel(g, b, 0, k_set<T>, gpu, N, (T)123);
      auto zero = cf.zero(gpu+start, (N-start));
      auto copy = cf.copy(cpu, gpu, N);
      kset.precede(zero);
      zero.precede(copy);
    });
    
    executor.run(taskflow).wait();

    for(int i=0; i<start; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)123) < 1e-4);
    }
    for(int i=start; i<N; ++i) {
      REQUIRE(std::fabs(cpu[i] - (T)0) < 1e-4);
    }
  }

  delete [] cpu;
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Zero.i8") {
  zero<int8_t>();
}

TEST_CASE("Zero.i16") {
  zero<int16_t>();
}

TEST_CASE("Zero.i32") {
  zero<int32_t>();
}

TEST_CASE("Zero.f32") {
  zero<float>();
}

// --------------------------------------------------------
// Testcase: Barrier
// --------------------------------------------------------
template <typename T>
void barrier() {

  const unsigned n = 1000;
  
  tf::Taskflow taskflow;
  tf::Executor executor;
  
  T* cpu = nullptr;
  T* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<T*>(std::calloc(n, sizeof(T)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&](tf::cudaFlow& cf) {

    dim3 g = {1, 1, 1};
    dim3 b = {1, 1, 1};
    auto br1 = cf.noop();
    auto br2 = cf.noop();
    auto br3 = cf.noop();
    auto h2d = cf.copy(gpu, cpu, n);
    auto d2h = cf.copy(cpu, gpu, n);

    h2d.precede(br1);

    for(unsigned i=0; i<n; ++i) {
      auto k1 = cf.kernel(g, b, 0, k_single_set<T>, gpu, i, (T)17);
      k1.succeed(br1)
        .precede(br2);

      auto k2 = cf.kernel(g, b, 0, k_single_add<T>, gpu, i, (T)3);
      k2.succeed(br2)
        .precede(br3);
    }

    br3.precede(d2h);
  });

  cputask.precede(gputask);
  
  executor.run(taskflow).wait();

  for(unsigned i=0; i<n; ++i) {
    REQUIRE(cpu[i] == (T)20);
  }

  std::free(cpu);
  REQUIRE(hipFree(gpu) == hipSuccess);
}

TEST_CASE("Barrier.i8" * doctest::timeout(300)) {
  barrier<int8_t>();
}

TEST_CASE("Barrier.i16" * doctest::timeout(300)) {
  barrier<int16_t>();
}

TEST_CASE("Barrier.i32" * doctest::timeout(300)) {
  barrier<int32_t>();
}

// ----------------------------------------------------------------------------
// NestedRuns
// ----------------------------------------------------------------------------

TEST_CASE("NestedRuns") {
  
  int* cpu = nullptr;
  int* gpu = nullptr;

  constexpr unsigned n = 1000;

  cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
  REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);

  struct A {

    tf::Executor executor;
    tf::Taskflow taskflow;

    void run(int* cpu, int* gpu, unsigned n) {
      taskflow.clear();

      auto A1 = taskflow.emplace([&](tf::cudaFlow& cf) {  
        cf.copy(gpu, cpu, n);
      });

      auto A2 = taskflow.emplace([&](tf::cudaFlow& cf) { 
        dim3 g = {(n+255)/256, 1, 1};
        dim3 b = {256, 1, 1};
        cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
      });

      auto A3 = taskflow.emplace([&] (tf::cudaFlow& cf) {
        cf.copy(cpu, gpu, n);
      });

      A1.precede(A2);
      A2.precede(A3);

      executor.run_n(taskflow, 10).wait();
    }

  };
  
  struct B {

    tf::Taskflow taskflow;
    tf::Executor executor;

    A a;

    void run(int* cpu, int* gpu, unsigned n) {

      taskflow.clear();
      
      auto B0 = taskflow.emplace([] () {});
      auto B1 = taskflow.emplace([&] (tf::cudaFlow& cf) { 
        dim3 g = {(n+255)/256, 1, 1};
        dim3 b = {256, 1, 1};
        auto h2d = cf.copy(gpu, cpu, n);
        auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
        auto d2h = cf.copy(cpu, gpu, n);
        h2d.precede(kernel);
        kernel.precede(d2h);
      });
      auto B2 = taskflow.emplace([&] () { a.run(cpu, gpu, n); });
      auto B3 = taskflow.emplace([&] (tf::cudaFlow&) { 
        for(unsigned i=0; i<n; ++i) {
          cpu[i]++;
        }
      });
      
      B0.precede(B1);
      B1.precede(B2);
      B2.precede(B3);

      executor.run_n(taskflow, 100).wait();
    }
  };

  B b;
  b.run(cpu, gpu, n);

  for(unsigned i=0; i<n; i++) {
    REQUIRE(cpu[i] == 1200);
  }
    
  REQUIRE(hipFree(gpu) == hipSuccess);
  std::free(cpu);
}

// ----------------------------------------------------------------------------
// WorkerID
// ----------------------------------------------------------------------------

void worker_id(unsigned N, unsigned M) {
  
  tf::Taskflow taskflow;
  tf::Executor executor(N, M);

  REQUIRE(executor.num_workers() == (N + M));
  REQUIRE(executor.num_domains() == 2);

  const unsigned s = 1000;

  for(unsigned k=0; k<s; ++k) {
    
    auto cputask = taskflow.emplace([&](){
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N);
    });
    
    auto gputask = taskflow.emplace([&](tf::cudaFlow&) {
      auto id = executor.this_worker_id();
      REQUIRE(id >= N);
      REQUIRE(id <  N+M);
    });

    auto chktask = taskflow.emplace([&] () {
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N);
    });
    
    taskflow.emplace([&](tf::cudaFlow&) {
      auto id = executor.this_worker_id();
      REQUIRE(id >= N);
      REQUIRE(id <  N+M);
    });
    
    taskflow.emplace([&]() {
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N);
    });

    auto subflow = taskflow.emplace([&](tf::Subflow& sf){
      auto id = executor.this_worker_id();
      REQUIRE(id >= 0);
      REQUIRE(id <  N);
      auto t1 = sf.emplace([&](){
        auto id = executor.this_worker_id();
        REQUIRE(id >= 0);
        REQUIRE(id <  N);
      });
      auto t2 = sf.emplace([&](tf::cudaFlow&){
        auto id = executor.this_worker_id();
        REQUIRE(id >= N);
        REQUIRE(id <  N+M);
      });
      t1.precede(t2);
    });

    cputask.precede(gputask);
    gputask.precede(chktask);
    chktask.precede(subflow);
  }

  executor.run_n(taskflow, 100).wait();
}

TEST_CASE("WorkerID.1C1G") {
  worker_id(1, 1);
}

TEST_CASE("WorkerID.1C2G") {
  worker_id(1, 2);
}

TEST_CASE("WorkerID.1C3G") {
  worker_id(1, 3);
}

TEST_CASE("WorkerID.1C4G") {
  worker_id(1, 4);
}

TEST_CASE("WorkerID.2C1G") {
  worker_id(2, 1);
}

TEST_CASE("WorkerID.2C2G") {
  worker_id(2, 2);
}

TEST_CASE("WorkerID.2C3G") {
  worker_id(2, 3);
}

TEST_CASE("WorkerID.2C4G") {
  worker_id(2, 4);
}

TEST_CASE("WorkerID.3C1G") {
  worker_id(3, 1);
}

TEST_CASE("WorkerID.3C2G") {
  worker_id(3, 2);
}

TEST_CASE("WorkerID.3C3G") {
  worker_id(3, 3);
}

TEST_CASE("WorkerID.3C4G") {
  worker_id(3, 4);
}

TEST_CASE("WorkerID.4C1G") {
  worker_id(4, 1);
}

TEST_CASE("WorkerID.4C2G") {
  worker_id(4, 2);
}

TEST_CASE("WorkerID.4C3G") {
  worker_id(4, 3);
}

TEST_CASE("WorkerID.4C4G") {
  worker_id(4, 4);
}

// ----------------------------------------------------------------------------
// Multiruns
// ----------------------------------------------------------------------------

void multiruns(unsigned N, unsigned M) {

  tf::Taskflow taskflow;
  tf::Executor executor(N, M);

  const unsigned n = 1000;
  const unsigned s = 1000;

  int *cpu[s] = {0};
  int *gpu[s] = {0};

  for(unsigned k=0; k<s; ++k) {
    
    int number = ::rand()%100;

    auto cputask = taskflow.emplace([&, k](){
      cpu[k] = static_cast<int*>(std::calloc(n, sizeof(int)));
      REQUIRE(hipMalloc(&gpu[k], n*sizeof(int)) == hipSuccess);
    });
    
    auto gputask = taskflow.emplace([&, k, number](tf::cudaFlow& cf) {
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu[k], cpu[k], n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu[k], n, number);
      auto d2h = cf.copy(cpu[k], gpu[k], n);
      h2d.precede(kernel);
      kernel.precede(d2h);
    });

    auto chktask = taskflow.emplace([&, k, number] () {
      for(unsigned i=0; i<n; ++i) {
        REQUIRE(cpu[k][i] == number);
      }
    });

    cputask.precede(gputask);
    gputask.precede(chktask);

  }

  executor.run(taskflow).wait();
}

TEST_CASE("Multiruns.1C1G") {
  multiruns(1, 1);
}

TEST_CASE("Multiruns.1C2G") {
  multiruns(1, 2);
}

TEST_CASE("Multiruns.1C3G") {
  multiruns(1, 3);
}

TEST_CASE("Multiruns.1C4G") {
  multiruns(1, 4);
}

TEST_CASE("Multiruns.2C1G") {
  multiruns(2, 1);
}

TEST_CASE("Multiruns.2C2G") {
  multiruns(2, 2);
}

TEST_CASE("Multiruns.2C3G") {
  multiruns(2, 3);
}

TEST_CASE("Multiruns.2C4G") {
  multiruns(2, 4);
}

TEST_CASE("Multiruns.3C1G") {
  multiruns(3, 1);
}

TEST_CASE("Multiruns.3C2G") {
  multiruns(3, 2);
}

TEST_CASE("Multiruns.3C3G") {
  multiruns(3, 3);
}

TEST_CASE("Multiruns.3C4G") {
  multiruns(3, 4);
}

TEST_CASE("Multiruns.4C1G") {
  multiruns(4, 1);
}

TEST_CASE("Multiruns.4C2G") {
  multiruns(4, 2);
}

TEST_CASE("Multiruns.4C3G") {
  multiruns(4, 3);
}

TEST_CASE("Multiruns.4C4G") {
  multiruns(4, 4);
}

// ----------------------------------------------------------------------------
// Subflow
// ----------------------------------------------------------------------------

TEST_CASE("Subflow") {

  tf::Taskflow taskflow;
  tf::Executor executor;
    
  int* cpu = nullptr;
  int* gpu = nullptr;
  
  const unsigned n = 1000;

  auto partask = taskflow.emplace([&](tf::Subflow& sf){

    auto cputask = sf.emplace([&](){
      cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    });
    
    auto gputask = sf.emplace([&](tf::cudaFlow& cf) {
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
    });

    cputask.precede(gputask);
  });
    
  auto chktask = taskflow.emplace([&](){
    for(unsigned i=0; i<n ;++i){
      REQUIRE(cpu[i] == 1);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  partask.precede(chktask);

  executor.run(taskflow).wait();

}

// ----------------------------------------------------------------------------
// NestedSubflow
// ----------------------------------------------------------------------------

TEST_CASE("NestedSubflow") {

  tf::Taskflow taskflow;
  tf::Executor executor;
    
  int* cpu = nullptr;
  int* gpu = nullptr;
  
  const unsigned n = 1000;
    
  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
  });

  auto partask = taskflow.emplace([&](tf::Subflow& sf){
    
    auto gputask1 = sf.emplace([&](tf::cudaFlow& cf) {
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
    });

    auto subtask1 = sf.emplace([&](tf::Subflow& sf) {
      auto gputask2 = sf.emplace([&](tf::cudaFlow& cf) {
        dim3 g = {(n+255)/256, 1, 1};
        dim3 b = {256, 1, 1};
        auto h2d = cf.copy(gpu, cpu, n);
        auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
        auto d2h = cf.copy(cpu, gpu, n);
        h2d.precede(kernel);
        kernel.precede(d2h);
      });
      
      auto subtask2 = sf.emplace([&](tf::Subflow& sf){
        sf.emplace([&](tf::cudaFlow& cf) {
          dim3 g = {(n+255)/256, 1, 1};
          dim3 b = {256, 1, 1};
          auto h2d = cf.copy(gpu, cpu, n);
          auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
          auto d2h = cf.copy(cpu, gpu, n);
          h2d.precede(kernel);
          kernel.precede(d2h);
        });
      });

      gputask2.precede(subtask2);
    });

    gputask1.precede(subtask1);
  });
    
  auto chktask = taskflow.emplace([&](){
    for(unsigned i=0; i<n ;++i){
      REQUIRE(cpu[i] == 3);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  partask.precede(chktask)
         .succeed(cputask);

  executor.run(taskflow).wait();

}

// ----------------------------------------------------------------------------
// DetachedSubflow
// ----------------------------------------------------------------------------

TEST_CASE("DetachedSubflow") {

  tf::Taskflow taskflow;
  tf::Executor executor;
    
  int* cpu = nullptr;
  int* gpu = nullptr;
  
  const unsigned n = 1000;

  taskflow.emplace([&](tf::Subflow& sf){

    auto cputask = sf.emplace([&](){
      cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
      REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    });
    
    auto gputask = sf.emplace([&](tf::cudaFlow& cf) {
      dim3 g = {(n+255)/256, 1, 1};
      dim3 b = {256, 1, 1};
      auto h2d = cf.copy(gpu, cpu, n);
      auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
      auto d2h = cf.copy(cpu, gpu, n);
      h2d.precede(kernel);
      kernel.precede(d2h);
    });

    cputask.precede(gputask);

    sf.detach();
  });
    
  executor.run(taskflow).wait();
  
  for(unsigned i=0; i<n ;++i){
    REQUIRE(cpu[i] == 1);
  }
  REQUIRE(hipFree(gpu) == hipSuccess);
  std::free(cpu);
}

// ----------------------------------------------------------------------------
// Conditional GPU tasking
// ----------------------------------------------------------------------------

TEST_CASE("Loop") {

  tf::Taskflow taskflow;
  tf::Executor executor;

  const unsigned n = 1000;
    
  int* cpu = nullptr;
  int* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&](tf::cudaFlow& cf) {
    dim3 g = {(n+255)/256, 1, 1};
    dim3 b = {256, 1, 1};
    auto h2d = cf.copy(gpu, cpu, n);
    auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
    auto d2h = cf.copy(cpu, gpu, n);
    h2d.precede(kernel);
    kernel.precede(d2h);
  });

  auto condition = taskflow.emplace([&cpu, round=0] () mutable {
    ++round;
    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == round);
    }
    return round >= 100;
  });

  auto freetask = taskflow.emplace([&](){
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  cputask.precede(gputask);
  gputask.precede(condition);
  condition.precede(gputask, freetask);
  
  executor.run(taskflow).wait();
}

// ----------------------------------------------------------------------------
// Predicate
// ----------------------------------------------------------------------------

TEST_CASE("Predicate") {

  tf::Taskflow taskflow;
  tf::Executor executor;

  const unsigned n = 1000;
    
  int* cpu = nullptr;
  int* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    REQUIRE(hipMemcpy(gpu, cpu, n*sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&](tf::cudaFlow& cf) {
    dim3 g = {(n+255)/256, 1, 1};
    dim3 b = {256, 1, 1};
    auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
    auto copy = cf.copy(cpu, gpu, n);
    kernel.precede(copy);
    cf.predicate([i=100]() mutable { return i-- == 0; });
  });

  auto freetask = taskflow.emplace([&](){
    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == 100);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  cputask.precede(gputask);
  gputask.precede(freetask);
  
  executor.run(taskflow).wait();
}

// ----------------------------------------------------------------------------
// Repeat
// ----------------------------------------------------------------------------

TEST_CASE("Repeat") {

  tf::Taskflow taskflow;
  tf::Executor executor;

  const unsigned n = 1000;
    
  int* cpu = nullptr;
  int* gpu = nullptr;

  auto cputask = taskflow.emplace([&](){
    cpu = static_cast<int*>(std::calloc(n, sizeof(int)));
    REQUIRE(hipMalloc(&gpu, n*sizeof(int)) == hipSuccess);
    REQUIRE(hipMemcpy(gpu, cpu, n*sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
  });

  auto gputask = taskflow.emplace([&](tf::cudaFlow& cf) {
    dim3 g = {(n+255)/256, 1, 1};
    dim3 b = {256, 1, 1};
    auto kernel = cf.kernel(g, b, 0, k_add<int>, gpu, n, 1);
    auto copy = cf.copy(cpu, gpu, n);
    kernel.precede(copy);
    cf.repeat(100);
  });

  auto freetask = taskflow.emplace([&](){
    for(unsigned i=0; i<n; ++i) {
      REQUIRE(cpu[i] == 100);
    }
    REQUIRE(hipFree(gpu) == hipSuccess);
    std::free(cpu);
  });

  cputask.precede(gputask);
  gputask.precede(freetask);
  
  executor.run(taskflow).wait();
}


